#include "hip/hip_runtime.h"
extern "C" {

#define TILE_WIDTH      ${tile_width}
#define KERNEL_WIDTH    3
#define KERNEL_RADIUS   1
#define CACHE_WIDTH     (TILE_WIDTH+KERNEL_WIDTH-1)  

__device__
float quadric(float norm, float thre) {
    return 1.f / (1.f + norm*norm / (thre*thre));
}

__device__
float exponential(float norm, float thre) {
    return exp(-norm*norm / (thre*thre));
}

__global__
void perona_malik_2d_kernel(
    float *dst,
    const float *src,
    const float thre, const float lambda,
    const int nx, const int ny
) {
    __shared__ 
    float cache[CACHE_WIDTH*CACHE_WIDTH];

    // cache linear index
    int ic = TILE_WIDTH*threadIdx.y + threadIdx.x;
    // load padded data to cache
    for (int i = ic; i < CACHE_WIDTH*CACHE_WIDTH; i += TILE_WIDTH*TILE_WIDTH) {
        // cache index
        int cx = i%CACHE_WIDTH;
        int cy = i/CACHE_WIDTH;

        // padded global index
        int pgx = TILE_WIDTH*blockIdx.x + cx - KERNEL_RADIUS;
        int pgy = TILE_WIDTH*blockIdx.y + cy - KERNEL_RADIUS;
        
        // mirror padding
        if ((pgx < 0) || (pgx >= nx) || (pgy < 0) || (pgy >= ny)) {
            cache[i] = 0.f;
        } else {
            cache[i] = src[nx*pgy+pgx];
        }
    }

    __syncthreads();

    // global index
    int gx = TILE_WIDTH*blockIdx.x + threadIdx.x;
    int gy = TILE_WIDTH*blockIdx.y + threadIdx.y;
    if ((gx >= nx) || (gy >= ny)) {
        return;
    }

    // local index
    int lx = threadIdx.x+KERNEL_RADIUS;
    int ly = threadIdx.y+KERNEL_RADIUS;

    // data grid
    //   x u x 
    //   l c r
    //   x d x
    float pc = cache[CACHE_WIDTH*ly + lx];
    float pu = cache[CACHE_WIDTH*(ly+1) + lx];
    float pd = cache[CACHE_WIDTH*(ly-1) + lx];
    float pr = cache[CACHE_WIDTH*ly + (lx+1)];
    float pl = cache[CACHE_WIDTH*ly + (lx-1)]; 

    // delta
    float du = pu-pc;
    float dd = pd-pc;
    float dr = pr-pc;
    float dl = pl-pc;

    // apply function
    // TODO use function pointer, assume quadric for now
    float cu = exponential(abs(du), thre);
    float cd = exponential(abs(dd), thre);
    float cr = exponential(abs(dr), thre);
    float cl = exponential(abs(dl), thre);

    // global linear index
    dst[nx*gy+gx] = pc + lambda * (cu*du + cd*dd + cr*dr + cl*dl);
}

}